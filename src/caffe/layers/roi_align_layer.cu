#include "hip/hip_runtime.h"
#include <cfloat>
#include <algorithm>
#include <vector>

#include "caffe/layers/roi_align_layer.hpp"
using std::max;
using std::min;

namespace roi_align_gpu {

	template <typename T>
	__device__ T bilinear_interpolate(
		const T* bottom_data,
		const int height,
		const int width,
		T y,
		T x,
		const int index /* index for debug only*/) {
		// deal with cases that inverse elements are out of feature map boundary
		if (y < -1.0 || y > height || x < -1.0 || x > width) {
			// empty
			return 0;
		}

		if (y <= 0) {
			y = 0;
		}
		if (x <= 0) {
			x = 0;
		}

		int y_low = (int)y;
		int x_low = (int)x;
		int y_high;
		int x_high;

		if (y_low >= height - 1) {
			y_high = y_low = height - 1;
			y = (T)y_low;
		}
		else {
			y_high = y_low + 1;
		}

		if (x_low >= width - 1) {
			x_high = x_low = width - 1;
			x = (T)x_low;
		}
		else {
			x_high = x_low + 1;
		}

		T ly = y - y_low;
		T lx = x - x_low;
		T hy = 1. - ly, hx = 1. - lx;
		// do bilinear interpolation
		T v1 = bottom_data[y_low * width + x_low];
		T v2 = bottom_data[y_low * width + x_high];
		T v3 = bottom_data[y_high * width + x_low];
		T v4 = bottom_data[y_high * width + x_high];
		T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

		T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

		return val;
	}

	template <typename T>
	__global__ void RoIAlignForward(
		const int nthreads,
		const T* bottom_data,
		const T spatial_scale,
		const int channels,
		const int height,
		const int width,
		const int pooled_height,
		const int pooled_width,
		const int sampling_ratio,
		const T* bottom_rois,
		int roi_cols,
		T* top_data) {
		CUDA_1D_KERNEL_LOOP(index, nthreads) {
			// (n, c, ph, pw) is an element in the pooled output
			int pw = index % pooled_width;
			int ph = (index / pooled_width) % pooled_height;
			int c = (index / pooled_width / pooled_height) % channels;
			int n = index / pooled_width / pooled_height / channels;

			// RoI could have 4 or 5 columns
			const T* offset_bottom_rois = bottom_rois + n * roi_cols;
			int roi_batch_ind = 0;
			if (roi_cols == 5) {
				roi_batch_ind = offset_bottom_rois[0];
				offset_bottom_rois++;
			}

			// Do not using rounding; this implementation detail is critical
			T roi_start_w = offset_bottom_rois[0] * spatial_scale;
			T roi_start_h = offset_bottom_rois[1] * spatial_scale;
			T roi_end_w = offset_bottom_rois[2] * spatial_scale;
			T roi_end_h = offset_bottom_rois[3] * spatial_scale;
			// T roi_start_w = roundf(offset_bottom_rois[0] * spatial_scale);
			// T roi_start_h = roundf(offset_bottom_rois[1] * spatial_scale);
			// T roi_end_w = roundf(offset_bottom_rois[2] * spatial_scale);
			// T roi_end_h = roundf(offset_bottom_rois[3] * spatial_scale);

			// Force malformed ROIs to be 1x1
			T roi_width = max(roi_end_w - roi_start_w, (T)1.);
			T roi_height = max(roi_end_h - roi_start_h, (T)1.);
			T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
			T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

			const T* offset_bottom_data =
				bottom_data + (roi_batch_ind * channels + c) * height * width;

			// We use roi_bin_grid to sample the grid and mimic integral
			int roi_bin_grid_h = (sampling_ratio > 0)
				? sampling_ratio
				: ceil(roi_height / pooled_height); // e.g., = 2
			int roi_bin_grid_w =
				(sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

			// We do average (integral) pooling inside a bin
			const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

			T output_val = 0.;
			for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
			{
				const T y = roi_start_h + ph * bin_size_h +
					static_cast<T>(iy + .5f) * bin_size_h /
					static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
				for (int ix = 0; ix < roi_bin_grid_w; ix++) {
					const T x = roi_start_w + pw * bin_size_w +
						static_cast<T>(ix + .5f) * bin_size_w /
						static_cast<T>(roi_bin_grid_w);

					T val = bilinear_interpolate(
						offset_bottom_data, height, width, y, x, index);
					output_val += val;
				}
			}
			output_val /= count;

			top_data[index] = output_val;
		}
	}

	//backward
	template <typename T>
	inline __device__ T gpu_atomic_add(const T val, T* address);

	template <>
	inline __device__ float gpu_atomic_add(const float val, float* address) {
		return atomicAdd(address, val);
	}
	template <>
	inline __device__ double gpu_atomic_add(const double val, double* address) {
		return atomicAdd(address, val);
	}

	template <typename T>
	__device__ void bilinear_interpolate_gradient(
		const int height,
		const int width,
		T y,
		T x,
		T& w1,
		T& w2,
		T& w3,
		T& w4,
		int& x_low,
		int& x_high,
		int& y_low,
		int& y_high,
		const int index /* index for debug only*/) {
		// deal with cases that inverse elements are out of feature map boundary
		if (y < -1.0 || y > height || x < -1.0 || x > width) {
			// empty
			w1 = w2 = w3 = w4 = 0.;
			x_low = x_high = y_low = y_high = -1;
			return;
		}

		if (y <= 0) {
			y = 0;
		}
		if (x <= 0) {
			x = 0;
		}

		y_low = (int)y;
		x_low = (int)x;

		if (y_low >= height - 1) {
			y_high = y_low = height - 1;
			y = (T)y_low;
		}
		else {
			y_high = y_low + 1;
		}

		if (x_low >= width - 1) {
			x_high = x_low = width - 1;
			x = (T)x_low;
		}
		else {
			x_high = x_low + 1;
		}

		T ly = y - y_low;
		T lx = x - x_low;
		T hy = 1. - ly, hx = 1. - lx;

		// reference in forward
		// T v1 = bottom_data[y_low * width + x_low];
		// T v2 = bottom_data[y_low * width + x_high];
		// T v3 = bottom_data[y_high * width + x_low];
		// T v4 = bottom_data[y_high * width + x_high];
		// T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

		w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

		return;
	}

	template <typename T>
	__global__ void RoIAlignBackwardFeature(
		const int nthreads,
		const T* top_diff,
		const int num_rois,
		const T spatial_scale,
		const int channels,
		const int height,
		const int width,
		const int pooled_height,
		const int pooled_width,
		const int sampling_ratio,
		T* bottom_diff,
		const T* bottom_rois) {
		CUDA_1D_KERNEL_LOOP(index, nthreads) {
			// (n, c, ph, pw) is an element in the pooled output
			int pw = index % pooled_width;
			int ph = (index / pooled_width) % pooled_height;
			int c = (index / pooled_width / pooled_height) % channels;
			int n = index / pooled_width / pooled_height / channels;

			const T* offset_bottom_rois = bottom_rois + n * 5;
			int roi_batch_ind = offset_bottom_rois[0];

			// Do not using rounding; this implementation detail is critical
			T roi_start_w = offset_bottom_rois[1] * spatial_scale;
			T roi_start_h = offset_bottom_rois[2] * spatial_scale;
			T roi_end_w = offset_bottom_rois[3] * spatial_scale;
			T roi_end_h = offset_bottom_rois[4] * spatial_scale;
			// T roi_start_w = roundf(offset_bottom_rois[1] * spatial_scale);
			// T roi_start_h = roundf(offset_bottom_rois[2] * spatial_scale);
			// T roi_end_w = roundf(offset_bottom_rois[3] * spatial_scale);
			// T roi_end_h = roundf(offset_bottom_rois[4] * spatial_scale);

			// Force malformed ROIs to be 1x1
			T roi_width = max(roi_end_w - roi_start_w, (T)1.);
			T roi_height = max(roi_end_h - roi_start_h, (T)1.);
			T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
			T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

			T* offset_bottom_diff =
				bottom_diff + (roi_batch_ind * channels + c) * height * width;

			int top_offset = (n * channels + c) * pooled_height * pooled_width;
			const T* offset_top_diff = top_diff + top_offset;
			const T top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

			// We use roi_bin_grid to sample the grid and mimic integral
			int roi_bin_grid_h = (sampling_ratio > 0)
				? sampling_ratio
				: ceil(roi_height / pooled_height); // e.g., = 2
			int roi_bin_grid_w =
				(sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

			// We do average (integral) pooling inside a bin
			const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

			for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
			{
				const T y = roi_start_h + ph * bin_size_h +
					static_cast<T>(iy + .5f) * bin_size_h /
					static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
				for (int ix = 0; ix < roi_bin_grid_w; ix++) {
					const T x = roi_start_w + pw * bin_size_w +
						static_cast<T>(ix + .5f) * bin_size_w /
						static_cast<T>(roi_bin_grid_w);

					T w1, w2, w3, w4;
					int x_low, x_high, y_low, y_high;

					bilinear_interpolate_gradient(
						height,
						width,
						y,
						x,
						w1,
						w2,
						w3,
						w4,
						x_low,
						x_high,
						y_low,
						y_high,
						index);

					T g1 = top_diff_this_bin * w1 / count;
					T g2 = top_diff_this_bin * w2 / count;
					T g3 = top_diff_this_bin * w3 / count;
					T g4 = top_diff_this_bin * w4 / count;

					if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
						gpu_atomic_add(
							static_cast<T>(g1), offset_bottom_diff + y_low * width + x_low);
						gpu_atomic_add(
							static_cast<T>(g2), offset_bottom_diff + y_low * width + x_high);
						gpu_atomic_add(
							static_cast<T>(g3), offset_bottom_diff + y_high * width + x_low);
						gpu_atomic_add(
							static_cast<T>(g4), offset_bottom_diff + y_high * width + x_high);
					} // if
				} // ix
			} // iy
		} // CUDA_1D_KERNEL_LOOP
	} // RoIAlignBackward

}

namespace caffe {
	
template <typename Dtype>
void ROIAlignLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_rois = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  const int nthreads = count;
  int batch_size = bottom[0]->num();
  int channels = bottom[0]->channels();
  int height = bottom[0]->height();
  int width = bottom[0]->width();
  int num_rois = bottom[1]->num();
  roi_align_gpu::RoIAlignForward<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS >>> (
	  nthreads,
	  bottom_data,
	  spatial_scale_,
	  channels,
	  height,
	  width,
	  pooled_height_,
	  pooled_width_,
	  sampling_ratio_,
	  bottom_rois,
	  num_rois,
	  top_data);
}



template <typename Dtype>
void ROIAlignLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* bottom_rois = bottom[1]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);

  //const int* argmax_data = max_idx_.gpu_data();
  int num_rois = bottom[1]->num();
  int batch_size = bottom[0]->num();
  int channels = bottom[0]->channels();
  int height = bottom[0]->height();
  int width = bottom[0]->width();
  const int nthreads = count;

  roi_align_gpu::RoIAlignBackwardFeature<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS >>> (
	  nthreads,
	  top_diff,
	  num_rois,
	  spatial_scale_,
	  channels,
	  height,
	  width,
	  pooled_height_,
	  pooled_width_,
	  sampling_ratio_,
	  bottom_diff,
	  bottom_rois
	  );
}

INSTANTIATE_LAYER_GPU_FUNCS(ROIAlignLayer);

}
