
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/cudnn_batch_norm_layer.hpp"
#include "caffe/util/math_functions.hpp"


namespace caffe {

	template<typename Dtype>
	void CuDNNBatchNormLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {

		const Dtype *bottom_data = bottom[0]->gpu_data();
		Dtype *top_data = top[0]->mutable_gpu_data();

		const Dtype *scale_data;
		const Dtype *bias_data;
		Dtype *global_mean;
		Dtype *global_var;
		Dtype *save_mean;
		Dtype *save_inv_var;

		if (this->phase_ == TRAIN && !use_global_stats_) {
			global_mean = this->blobs_[0]->mutable_gpu_data();
			global_var = this->blobs_[1]->mutable_gpu_data();
			save_mean = save_mean_.mutable_gpu_data();
			save_inv_var = save_inv_var_.mutable_gpu_data();
		}
		else {
			global_mean = this->blobs_[0]->mutable_gpu_data();
			global_var = this->blobs_[1]->mutable_gpu_data();
		}
		if (this->scale_bias_) {
			scale_data = this->blobs_[3]->gpu_data();
			bias_data = this->blobs_[4]->gpu_data();
		}
		else {
			scale_data = scale_ones_.gpu_data();
			bias_data = bias_zeros_.gpu_data();
		}

		if (this->phase_ == TRAIN ) {
			Dtype factor = 1. - this->moving_average_fraction_;
			if (use_global_stats_) {
				factor = 0;
			}
			CUDNN_CHECK(cudnnBatchNormalizationForwardTraining(handle_, mode_,
				cudnn::dataType<Dtype>::one, cudnn::dataType<Dtype>::zero,
				bottom_desc_, bottom_data, top_desc_, top_data,
				scale_bias_mean_var_desc_, scale_data, bias_data,
				factor, global_mean, global_var, CUDNN_BN_MIN_EPSILON, save_mean, save_inv_var));
		}
		else if (this->phase_ == TEST) {
			CUDNN_CHECK(cudnnBatchNormalizationForwardInference(handle_,
				CUDNN_BATCHNORM_SPATIAL,
				cudnn::dataType<Dtype>::one, cudnn::dataType<Dtype>::zero,
				bottom_desc_, bottom_data, top_desc_, top_data,
				scale_bias_mean_var_desc_, scale_data, bias_data,
				global_mean, global_var, CUDNN_BN_MIN_EPSILON));
		}
		else {
			LOG(FATAL) << "Unknown phase";
		}
		//CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream()));
	}


	template <typename Dtype>
	void CuDNNBatchNormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

		const Dtype *top_diff = top[0]->gpu_diff();
		Dtype * bottom_diff = bottom[0]->mutable_gpu_diff();
		const Dtype *bottom_data = bottom[0]->gpu_data();
		double epsilon = this->eps_;
		const Dtype* save_mean;
		const Dtype* save_inv_var;
		const Dtype* scale_data;
		Dtype*  scale_diff;
		Dtype*  bias_diff;

		save_mean = save_mean_.gpu_data();
		save_inv_var = save_inv_var_.gpu_data();
		if (this->scale_bias_) {
			scale_data = this->blobs_[3]->gpu_data();
			scale_diff = this->blobs_[3]->mutable_gpu_diff();
			bias_diff = this->blobs_[4]->mutable_gpu_diff();
		}
		else {
			scale_data = scale_ones_.gpu_data();
			scale_diff = scale_ones_.mutable_gpu_diff();
			bias_diff = bias_zeros_.mutable_gpu_diff();
		}

		CUDNN_CHECK(cudnnBatchNormalizationBackward(handle_, mode_,
			cudnn::dataType<Dtype>::one, cudnn::dataType<Dtype>::zero,
			cudnn::dataType<Dtype>::one, cudnn::dataType<Dtype>::one,
			bottom_desc_, bottom_data, bottom_desc_, top_diff, bottom_desc_, bottom_diff,
			scale_bias_mean_var_desc_, scale_data, scale_diff, bias_diff,
			CUDNN_BN_MIN_EPSILON, save_mean, save_inv_var));
		
		/*CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream(0)));*/
	}

	INSTANTIATE_LAYER_GPU_FUNCS(CuDNNBatchNormLayer);

}

#endif