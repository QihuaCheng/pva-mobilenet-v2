
#include "caffe/layers/syshen_convolution.h"


template <typename Dtype>
syshen_convolution<Dtype>::syshen_convolution() {
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&input_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&output_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateFilterDescriptor(&filter_desc));
	CHECK_CUDNN_ERROR(hipdnnCreateConvolutionDescriptor(&conv_desc));
	if (has_bias) {
		CHECK_CUDNN_ERROR(hipdnnCreateTensorDescriptor(&bias));
	}
	CHECK_CUDNN_ERROR(hipdnnCreate(&handle_t));
	if (use_stream) {
		CHECK_CUDA_ERROR(hipStreamCreate(&stream));
		CHECK_CUDA_ERROR(hipEventCreate(&start));
	}
	batch = 1;
	in_channels = 1;
	stride_h = 1;
	stride_w = 1;
	pad_h = 1;
	pad_w = 1;
	dilation_h = 1;
	dilation_w = 1;
	kernel_h = 1;
	kernel_w = 1;
}

template <typename Dtype>
syshen_convolution<Dtype>::~syshen_convolution() {
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(input_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(output_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyFilterDescriptor(filter_desc));
	CHECK_CUDNN_ERROR(hipdnnDestroyConvolutionDescriptor(&conv_desc));
	if (has_bias) {
		CHECK_CUDNN_ERROR(hipdnnDestroyTensorDescriptor(bias));
	}
	CHECK_CUDNN_ERROR(hipdnnDestroy(&handle_t));
	if (use_stream) {
		CHECK_CUDA_ERROR(hipStreamDestroy(stream));
		CHECK_CUDA_ERROR(hipEventDestroy(strat));
	}
}

template<typename Dtype>
void syshen_convolution<Dtype>::SetUp() {
	int nStride = in_channels * in_h * in_w;
	int cStride = in_h * in_w;

	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptorEx(
		input_desc, 
		hipdnnDataType_t::HIPDNN_DATA_FLOAT, 
		batch, 
		in_channels,
		in_h, in_w, nStride, cStride, in_w, 1));

	CHECK_CUDNN_ERROR(hipdnnSetFilter4dDescriptor(
		filter_dsec,
		hipdnnDataType_t::HIPDNN_DATA_FLOAT, 
		HIPDNN_TENSOR_NCHW, 
		out_channels, in_channels, kernel_h, kernel_w));

	CHECK_CUDNN_ERROR(hipdnnSetConvolution2dDescriptor(
		conv_desc, pad_h, pad_w, stride_h, 
		stride_w, dilation_h, dilation_w, 
		HIPDNN_CROSS_CORRELATION, hipdnnDataType_t::HIPDNN_DATA_FLOAT));

	/*CHECK_CUDNN_ERROR(hipdnnGetConvolution2dForwardOutputDim(
		conv_desc, input_desc, filter_dsec,
		&out_batch, &out_channels, &out_h, &out_w));*/

	CHECK_CUDNN_ERROR(hipdnnSetTensor4dDescriptor(
		output_desc, HIPDNN_TENSOR_NCHW,
		hipdnnDataType_t::HIPDNN_DATA_FLOAT, 
		out_batch, out_channels, out_h, out_w));

	CHECK_CUDNN_ERROR(hipdnnGetConvolutionForwardAlgorithm(
		handle_t, input_desc, filter_dsec, 
		conv_desc, output_desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
		0, &algo));

	CHECK_CUDNN_ERROR(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
		handle_t, input_desc, filter_dsec,
		conv_desc, output_desc, algo, &workSpaceSize));
	if (0 != workSpaceSize)
		CHECK_CUDA_ERROR(hipMalloc((void**)&workSpace, workSpaceSize));

	if (has_bias) {
		hipdnnSetTensor4dDescriptor(bias, HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, out_batch, out_channels, 1, 1);
	}
}